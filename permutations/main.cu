
#include <hip/hip_runtime.h>
#include <stdio.h>
/*

   This program computes the max number of inversion for a permutation of length n
   max permutations is (n^2 - n)/2

*/

__global__ void inversionPairs(int* row, int* inversions) {
    if (blockIdx.x < threadIdx.x) {
        if (row[blockIdx.x] > row[threadIdx.x]) {
            inversions[blockIdx.x*blockDim.x + threadIdx.x] = 1;
        }
    }
}

__global__ void countInversions(int* inversions, int width) {
    int sum = 0;
    for (int i = 0; i < width; i++) {
        sum += inversions[width*threadIdx.x+i];
        inversions[width*threadIdx.x+i] = 0;
    }
    inversions[width*threadIdx.x] = sum;

}

int main() {
    int n = 1<<10;

    // Allocate row
    int* host_row = (int*) malloc(sizeof(int)*n);
    for (int i = 0; i < n; i++) {
        host_row[n-1-i] = i;
    }
    int* dev_row;
    hipMalloc((void**)&dev_row, sizeof(int)*n);
    hipMemcpy(dev_row, host_row, sizeof(int)*n, hipMemcpyHostToDevice);

    // Allocate Inversions
    int* dev_inversions;
    hipMalloc((void**)&dev_inversions, sizeof(int)*n*n);
    int* host_inversions = (int*) malloc(sizeof(int)*n*n);

    // GPU funcs
    inversionPairs<<<n,n>>>(dev_row, dev_inversions);
    for (int i = 1; i <= n*n; i *= 2) {
        countInversions<<<1,(n*n)/i>>>(dev_inversions, i);
    }

    // Copy memory to host
    hipMemcpy(host_inversions, dev_inversions, sizeof(int)*n*n, hipMemcpyDeviceToHost);

    // Print
    /*
    for (int i = 0; i < n*n; i++) {
        if (host_inversions[i] == 1) {
            printf("(%d, %d)\n", i/n, i%n);
        }
    }
    */
    printf("Inversions: %d\n", host_inversions[0]);


    // Free memory
    free(host_row);
    free(host_inversions);
    hipFree(dev_row);
    hipFree(dev_inversions);


    return 0;
}
